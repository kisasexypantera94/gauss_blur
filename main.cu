#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <vector>
#include <exception>

// =========================================================================
texture<uchar4, 2> tex_in;
texture<uchar4, 2> tex_out;

__device__ void swap(int *x, int *y) {
    int tmp = *x;
    *x = *y;
    *y = tmp;
}

__global__ void gauss_blur(const int height,
                           const int width,
                           const double *const filter,
                           const int R,
                           uchar4 *const out,
                           const bool horizontal) {
    for (int x = threadIdx.x + blockDim.x * blockIdx.x; x < width; x += blockDim.x * gridDim.x) {
        for (int y = threadIdx.y + blockDim.y * blockIdx.y; y < height; y += blockDim.y * gridDim.y) {
            uchar4 weighted_sum = make_uchar4(0, 0, 0, 255);
            for (int r = -R; r <= R; ++r) {
                const int tex_x = x + (horizontal ? r : 0);
                const int tex_y = y + (horizontal ? 0 : r);

                const uchar4 tex = tex2D(horizontal ? tex_out : tex_in, tex_x, tex_y);
                const double scale = filter[abs(r)];
                weighted_sum.x += scale * tex.x;
                weighted_sum.y += scale * tex.y;
                weighted_sum.z += scale * tex.z;
            }

            out[width * y + x] = weighted_sum;
        }
    }
}

// =========================================================================
inline auto f(const double x, const double sigma) -> double {
    return 1 / sqrt(2 * M_PI * sigma*sigma) * exp(-x*x / (2 * sigma*sigma));
}

auto comp_filter(const size_t R) -> std::vector<double> {
    const double sigma = R / 3.0;

    std::vector<double> filter(R + 1);
    double sum = 0;
    for (size_t t = 1; t <= R; ++t) {
        filter[t] = f(t, sigma);
        sum += filter[t];
    }
    filter[0] = 1.0 - 2 * sum;

    return filter;
}

struct Image : std::vector<uchar4> {
    int height;
    int width;

    Image() = default;
    void load(const std::string &filename) {
        {
            std::ifstream ifs(filename, std::ios::binary | std::ios::in);
            if (!ifs.is_open()) {
                std::throw_with_nested(std::runtime_error("could not open file " + filename));
            }

            ifs.read(reinterpret_cast<char *>(&width), sizeof(int));
            ifs.read(reinterpret_cast<char *>(&height), sizeof(int));
            this->resize(height * width);
            ifs.read(reinterpret_cast<char *>(this->data()), height * width * sizeof(uchar4));
        }
    }

    void save(const std::string &filename) {
        std::ofstream ofs(filename, std::ios::binary | std::ios::out);
        if (!ofs.is_open()) {
            std::throw_with_nested(std::runtime_error("could not open file " + filename));
        }

        ofs.write(reinterpret_cast<const char *>(&width), sizeof(int));
        ofs.write(reinterpret_cast<const char *>(&height), sizeof(int));
        ofs.write(reinterpret_cast<const char *>(this->data()), height * width * sizeof(uchar4));
    }
};

inline void CHECK_ERR(hipError_t err) {
    if (err != hipSuccess) {
        printf("ERROR: %s\n", hipGetErrorString(err));
        exit(0);
    }
}

auto gauss_blur_image(const Image &h_in, const size_t R) -> Image {
    // Prepare filter
    const auto filter = comp_filter(R);

    // Prepare device buffers
    uchar4 *d_out;
    double *d_filter;
    CHECK_ERR(hipMalloc(&d_out, h_in.size() * sizeof(uchar4)));
    CHECK_ERR(hipMalloc(&d_filter, filter.size() * sizeof(double)));

    // Copy filter to device
    CHECK_ERR(hipMemcpy(d_filter, filter.data(), filter.size() * sizeof(double), hipMemcpyHostToDevice));

    // Bind textures to device buffers
    hipArray *arr_in;
    hipArray *arr_out;
    hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
    CHECK_ERR(hipMallocArray(&arr_in, &desc, h_in.width, h_in.height));
    CHECK_ERR(hipMallocArray(&arr_out, &desc, h_in.width, h_in.height));
    CHECK_ERR(hipMemcpyToArray(arr_in, 0, 0, h_in.data(), h_in.size() * sizeof(uchar4), hipMemcpyHostToDevice));

    CHECK_ERR(hipBindTextureToArray(tex_in, arr_in, desc));
    CHECK_ERR(hipBindTextureToArray(tex_out, arr_out, desc));

    // Run kernel
    auto block_dim = dim3(32, 32);
    auto grid_dim = dim3(32, 32);
    // Vertical
    gauss_blur<<<grid_dim, block_dim>>>(h_in.height, h_in.width, d_filter, R, d_out, false);
    CHECK_ERR(hipDeviceSynchronize());
    CHECK_ERR(hipGetLastError());

    CHECK_ERR(hipMemcpyToArray(arr_out, 0, 0, d_out, h_in.size() * sizeof(uchar4), hipMemcpyDeviceToDevice));
    // Horizontal
    gauss_blur<<<grid_dim, block_dim>>>(h_in.height, h_in.width, d_filter, R, d_out, true);
    CHECK_ERR(hipDeviceSynchronize());
    CHECK_ERR(hipGetLastError());

    // Get results
    Image h_out;
    h_out.resize(h_in.size());
    h_out.height = h_in.height;
    h_out.width = h_in.width;
    CHECK_ERR(hipMemcpy(h_out.data(), d_out, h_in.size() * sizeof(uchar4), hipMemcpyDeviceToHost));

    CHECK_ERR(hipUnbindTexture(tex_in));
    CHECK_ERR(hipUnbindTexture(tex_out));

    CHECK_ERR(hipFreeArray(arr_in));
    CHECK_ERR(hipFreeArray(arr_out));
    CHECK_ERR(hipFree(d_out));
    CHECK_ERR(hipFree(d_filter));

    return h_out;
}

int main() {
    std::string in;
    std::string out;
    size_t R;

    std::cin >> in;
    std::cin >> out;
    std::cin >> R;

    Image h_in;
    try {
        h_in.load(in);
    } catch (const std::exception &e) {
        std::cerr << "ERROR: " + std::string(e.what()) << std::endl;
        return 1;
    }

    auto h_out = gauss_blur_image(h_in, R);
    h_out.save(out);
}
